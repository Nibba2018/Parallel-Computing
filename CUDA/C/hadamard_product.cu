#include <stdio.h>
#include <hip/hip_runtime.h>


#define LOG_INPUT if(0)
#define LOG_OUTPUT if(1)
#define LOG if(0)


__global__ void hadamard(float *A, float *B, float *C, int M, int N)
{
    // Complete the kernel code snippet
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if(i < M*N)
    	C[i] = A[i]*B[i];
}

/**
 * Host main routine
 */
void print_matrix(float *A,int m,int n)
{
    for(int i =0;i<m;i++)
    {
        for(int j=0;j<n;j++)
            printf("%.2f ",A[i*n+j]);
        printf("\n");
    }

}
int main(void)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    
    int t; //number of test cases
    scanf("%d",&t);
    while(t--)
    {
        int m,n;
        scanf("%d %d",&m,&n);
        size_t size = m*n * sizeof(float);
        LOG printf("[Hadamard product of two matrices ]\n");

        // Allocate the host input vector A
        float *h_A = (float*)malloc(size);
        // Allocate the host input vector B
        float *h_B = (float*)malloc(size);
        // Allocate the host output vector C
        float *h_C = (float*)malloc(size);

        // Verify that allocations succeeded
        if (h_A == NULL || h_B == NULL || h_C == NULL)
        {
            fprintf(stderr, "Failed to allocate host vectors!\n");
            exit(EXIT_FAILURE);
        }

        // Initialize the host input vectors
        
        for (int i = 0; i < n*m; ++i)
        {
            scanf("%f",&h_A[i]);
            scanf("%f",&h_B[i]);

        }
        
        float *d_A = NULL, *d_B = NULL, *d_C = NULL; 
        // Allocate the device input vector A
        hipMalloc((void**)&d_A, size);

        // Allocate the device input vector B
        hipMalloc((void**)&d_B, size);

        // Allocate the device output vector C
        hipMalloc((void**)&d_C, size);

        // Copy the host input vectors A and B in host memory to the device input vectors in
        // device memory
		hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
		hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
		hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);        
        // initialize blocksPerGrid and threads Per Block

		int threadsPerBlock = 256;
    	int blocksPerGrid = ((m*n)+threadsPerBlock-1)/threadsPerBlock;
    	
        hadamard<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, m, n);
        err = hipGetLastError();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Copy the device result vector in device memory to the host result vector
        // in host memory.
        
        hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
        
        // Verify that the result vector is correct
        for (int i = 0; i < n*m; ++i)
        {
            if (fabs(h_A[i] * h_B[i] - h_C[i]) > 1e-5)
            {
                fprintf(stderr, "Result verification failed at element %d!\n", i);
                exit(EXIT_FAILURE);
            }
        }

        LOG printf("Test PASSED\n");

        // Free device global memory
        

        // Free host memory
        
        
        err = hipDeviceReset();

        if (err != hipSuccess)
        {
            fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
        print_matrix(h_C,m,n);
        
        LOG printf("Done\n");
    }
    return 0;
}

